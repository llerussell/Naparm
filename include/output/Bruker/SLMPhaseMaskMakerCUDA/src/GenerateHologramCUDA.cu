/*
    Hologram generating algorithms for CUDA Devices
    Copyright 2009, 2010, 2011, 2012 Martin Persson
    martin.persson@physics.gu.se
    Small edits by Lloyd Russell 2016

    This file is part of GenerateHologramCUDA.
    GenerateHologramCUDA is free software: you can redistribute it and/or
    modify it under the terms of the GNU Lesser General Public License as published
    by the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GenerateHologramCUDA is distributed in the hope that it will be
    useful, but WITHOUT ANY WARRANTY; without even the implied warranty
    of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU Lesser General Public License for more details.
    You should have received a copy of the GNU Lesser General Public License
    along with GenerateHologramCUDA.  If not, see <http://www.gnu.org/licenses/>.


    The function "GenerateHologram" contains two different algorithms for
    hologram generation. The last parameter in the function call selects which
    one to use:
    0: Complex addition of "Lenses and Prisms", no optimization (3D)
    1: Weighted Gerchberg-Saxton algorithm using Fresnel propagation (3D)
    2: Weighted Gerchberg-Saxton algorithm using Fast Fourier Transforms (2D)
        (0) produces optimal holograms for 1 or 2 traps and is significantly faster.
        (0) is automatically selected if the number of spots is < 3.

    Fresnel propagation based algorithm (1) described in:
    Roberto Di Leonardo, Francesca Ianni, and Giancarlo Ruocco
    "Computer generation of optimal holograms for optical trap arrays"
    Opt. Express 15, 1913-1922 (2007)
    The original algorithm has been modified to allow variable spot amplitudes

    Naming convention for variables:
    The prefix indicates where data is located
    In host functions:      h = host memory
                            d = device memory
                            c = constant memory
    In global functions:    g = global memory
                            s = shared memory
                            c = constant memory
                            no prefix = registers
    The suffix indicates the data type, no suffix usually indicates an integer

    Possible improvements:
    * Improve convergence of the GS algorithms for 2 spots.                         *done
    * Compensate spot intensities for distance from center of field.                *done
    * Put all arguments for device functions and trap positions in constant memory. *done
        (Requires all functions to be moved into the same file or the use of some
        workaround found on nVidia forum)
    * Put pSLMstart and aLaser in texture memory (may not improve performance on Fermi devices)
    * Use "zero-copy" to transfer pSLM to host.
    * Rename functions and variables for consistency and readability
    * Allow variable spot phases for Lenses and Prisms
*/

//#define M_CUDA_DEBUG			   //activates a number of custom debug macros//
float dt_milliseconds;
hipEvent_t start, stop;

//Includes

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include "stdint.h"
#include <string.h>
#include <math.h>
#include <hipfft/hipfft.h>

#ifndef M_PI
    #define M_PI 3.14159265358979323846f
#endif
#define MAX_SPOTS 1024	//decrease this if your GPU keeps running out of memory
#define BLOCK_SIZE 256	//should be a power of 2
#define SLM_SIZE 512
#if ((SLM_SIZE==16)||(SLM_SIZE==32)||(SLM_SIZE==64)||(SLM_SIZE==128)||(SLM_SIZE==256)||(SLM_SIZE==512)||(SLM_SIZE==1024)||(SLM_SIZE==2048))
    #define SLMPOW2			//Uses bitwize modulu operations if teh SLM size is a power of 2
#endif

// forward declarations
__global__ void ApplyCorrections(uint16_t *g_pSLM_uc, uint16_t *g_LUT, float *d_AberrationCorr_f, float *d_LUTPolCoeff_f);
__global__ void LensesAndPrisms(uint16_t *g_SLMuc, uint16_t *g_LUT, float *d_AberrationCorr_f, float *d_LUTPolCoeff_f);
__global__ void calculateIobtained(uint16_t *g_pSLM_uc, float *g_Iobtained);
__global__ void PropagateToSLM_Fresnel(float *g_spotRe_f, float *g_spotIm_f, float *g_pSLM2pi, float *g_weights, int iteration, float *g_pSLMstart, float *g_amps,
										bool getpSLM65535, uint16_t *g_pSLM65535_uc, uint16_t *g_LUT, float *g_AberrationCorr_f, float *g_LUTPolCoeff_f);
__global__ void PropagateToSLMDC_Fresnel(float *g_pSpot, float *g_wSpot, hipfftComplex *g_cSLM_cc, float *g_pSLM_f, int iteration, float *g_pSLMstart, bool getpSLM65535,
										 uint16_t *g_pSLM65535_uc);
__global__ void setActiveRegionToZero(hipfftComplex *g_Farfield);
__global__ void PropagateToSpotPositions_Fresnel(float *g_pSLM2pi, float *g_spotRe_f, float *g_spotIm_f);
__global__ void PropagateToSpotPositionsDC_Fresnel(float *g_pSLM_f, float *g_obtainedPhase, float *g_weights, float *g_Iobtained, int iteration);
__global__ void ReplaceAmpsSLM_FFT(float *g_aLaser, hipfftComplex *g_cAmp, float *g_pSLMstart, bool getpSLM65535, uint16_t *g_pSLM65535_uc, uint16_t *g_LUT, float *g_AberrationCorr_f, float *g_LUTPolCoeff_f);
__global__ void ReplaceAmpsSpots_FFT(hipfftComplex *g_cSpotAmp_cc, hipfftComplex *g_cSpotAmpNew_cc, int iteration, float *g_Iobtained, float *g_weight, bool last_iteration);
__global__ void ReplaceAmpsSpotsDC_FFT(hipfftComplex *g_cSpotAmp_cc, hipfftComplex *g_cSpotAmpNew_cc, int iteration, float *g_Iobtained, float *g_weight, bool last_iteration);
__global__ void XYtoIndex();
__global__ void f2uc(uint16_t *uc, float *f, int N_pixels, uint16_t *g_LUT, int use_linLUT, int data_w);
__global__ void uc2f(float *f, uint16_t *uc, int N);
__global__ void p2c(hipfftComplex *g_c, float *g_p, int M);
inline int computeAndCopySpotData(float *h_I, float *x, float *y, float *z, int N_spots, int method);

// Custom debug macros
#define M_CHECK_ERROR() mCheckError(__LINE__, __FILE__)
#define M_SAFE_CALL(errcode) mSafeCall(errcode, __LINE__, __FILE__)
#define M_CUFFT_SAFE_CALL(cuffterror) mCufftSafeCall(cuffterror, __LINE__, __FILE__)
#define M_DISPLAY_DATA_F(data, length) mDisplayDataF(data, length, __LINE__)
#define M_DISPLAY_DATA_UC(data, length) mDisplayDataUC(data, length, __LINE__)
#define M_DISPLAY_DATA_CC(data, length) mDisplayDataCC(data, length, __LINE__)
#define M_DISPLAY_DATA_I(data, length) mDisplayDataI(data, length, __LINE__)
inline void mSafeCall(hipError_t status, int line, const char *file);
inline void mCufftSafeCall(hipfftResult_t status, int line, const char *file);
inline void mCheckError(int line, const char *file);
inline void mDisplayDataF(float *d_data, int length, int line);
inline void mDisplayDataCC(hipfftComplex *d_data, int length, int line);
inline void mDisplayDataUC(uint16_t *d_data, int length, int line);
inline void mDisplayDataI(int *d_data, int length, int line);

//Global declaration
float *d_x, *d_y, *d_z, *d_I;					//trap coordinates and intensity in GPU memory
float *d_pSLM_f;								//the optimized pSpot pattern, float [-pi, pi]
float *d_weights, *d_Iobtained, *d_desiredAmp;	//used h_weights and calculated amplitudes for each spot and each iteration
float *d_pSLMstart_f;							//Initial pSpot pattern [-pi, pi]
float *d_spotRe_f, *d_spotIm_f;
float *d_AberrationCorr_f = NULL;
float *d_LUTPolCoeff_f = NULL;
float SLMsizef = (float)SLM_SIZE;
int N_PolLUTCoeff = 0;
int n_blocks_Phi, memsize_SLM_f, memsize_SLMuc, memsize_spotsf, data_w, N_pixels, N_iterations_last;
float h_desiredAmp[MAX_SPOTS];
int h_spotIndex[MAX_SPOTS];
uint16_t *d_pSLM_uc;						//The optimized pSpot pattern, uint16_t, the one sent to the SLM [0, 65535]
uint16_t *h_LUT_uc;
uint16_t *d_LUT_uc = NULL;
int maxThreads_device;
bool ApplyLUT_b = false, EnableSLM_b = false, UseAberrationCorr_b = false, UsePolLUT_b = false, saveI_b = false, useRPC_b = false, useDC_b = false;
float alphaRPC_f = 10;
char CUDAmessage[100];
hipError_t status;
float *d_aLaserFFT, *d_LUT_coeff;
hipfftHandle plan;
hipfftComplex *d_FFTo_cc, *d_FFTd_cc, *d_SLM_cc;
int *d_spot_index, memsize_SLMcc;
int borderWidthDC_i;
float *d_obtainedPhase;

//Constant memory declarations
__device__ __constant__ int c_data_w[1];
__device__ __constant__ float c_data_w_f[1];
__device__ __constant__ int c_half_w[1];
__device__ __constant__ float c_half_w_f[1];
__device__ __constant__ int c_N_pixels[1];
__device__ __constant__ float c_N_pixels_f[1];
__device__ __constant__ float c_SLMpitch_f[1];
__device__ __constant__ bool c_useDC_b[1];
__device__ __constant__ int c_DCborderWidth[1];
__device__ __constant__ bool c_useRPC_b[1];
__device__ __constant__ float c_alphaRPC_f[1];
__device__ __constant__ bool c_saveI_b[1];
__device__ __constant__ int c_log2data_w[1];
__device__ __constant__ float c_x[MAX_SPOTS];
__device__ __constant__ float c_y[MAX_SPOTS];
__device__ __constant__ float c_z[MAX_SPOTS];
__device__ __constant__ float c_desiredAmp[MAX_SPOTS];
__device__ __constant__ int c_spotIndex[MAX_SPOTS];
__device__ __constant__ int c_N_spots[1];

//Public dll functions
//Generate a hologram
extern "C" __declspec(dllexport)  int GenerateHologram(float *h_checkData, uint16_t *h_pSLM_uc, float *x_spots, float *y_spots, float *z_spots, float *I_spots, int N_spots, int N_iterations, float *h_Iobtained, int method)//, float* gpuTime)
{
	//*gpuTime = 0;
	//float deltaTime = 0;
	if (N_spots > MAX_SPOTS)
		N_spots = MAX_SPOTS;
	else if (N_spots < 1)
		method = 100;
	else if (N_spots < 3)
		method = 0;
	memsize_spotsf = N_spots*sizeof(float);
	method = computeAndCopySpotData(I_spots, x_spots, y_spots, z_spots, N_spots, method); //sets method to -1 if N_spots == 0.
	switch (method)	{
		case 0:
			//////
			//Generate the hologram using "Lenses and Prisms"
			//////
			LensesAndPrisms<<< n_blocks_Phi, BLOCK_SIZE >>>(d_pSLM_uc, d_LUT_uc, d_AberrationCorr_f, d_LUTPolCoeff_f);
			M_CHECK_ERROR();
			hipDeviceSynchronize();
			M_CHECK_ERROR();
			if (saveI_b)
			{
				calculateIobtained<<< N_spots, SLM_SIZE>>>(d_pSLM_uc, d_Iobtained);
				M_CHECK_ERROR();
				hipDeviceSynchronize();
				M_SAFE_CALL(hipMemcpy(h_Iobtained, d_Iobtained, N_spots*sizeof(float), hipMemcpyDeviceToHost));
			}
			M_SAFE_CALL(hipMemcpy(h_pSLM_uc, d_pSLM_uc, memsize_SLMuc, hipMemcpyDeviceToHost));
			break;
		case 1:
			//Generate holgram using fresnel propagation
			//Uncomment this to start with pre-calculated hologram:
			//cudaMemcpy(d_pSLM_uc, h_pSLM_uc, memsize_SLMuc, cudaMemcpyHostToDevice);
			//cudaDeviceSynchronize();
			//uc2f<<< n_blocks_Phi, BLOCK_SIZE >>>(d_pSLM_f, d_pSLM_uc, N_pixels);
			/*cudaEventCreate(&start);
			cudaEventCreate(&stop);
			cudaEventRecord(start, 0);
			cudaEventSynchronize(start);*/
			for (int l=0; l<N_iterations; l++)
			{
				//Propagate to the spot positions
				if (useDC_b)
				{
					M_CUFFT_SAFE_CALL(hipfftExecC2C(plan, d_SLM_cc, d_FFTo_cc, HIPFFT_FORWARD));
					M_CHECK_ERROR();
					PropagateToSpotPositionsDC_Fresnel<<< N_spots, SLM_SIZE>>>(d_pSLM_f, d_obtainedPhase, d_weights, d_Iobtained, l);  //this function is very slow
					M_CHECK_ERROR();
					setActiveRegionToZero<<< SLM_SIZE, SLM_SIZE >>>(d_FFTo_cc);
				}
				else
					PropagateToSpotPositions_Fresnel<<< N_spots, SLM_SIZE>>>(d_pSLM_f, d_spotRe_f, d_spotIm_f);
				M_CHECK_ERROR();
				hipDeviceSynchronize();

				//Propagate to the SLM plane
				if (useDC_b)
				{
					M_CUFFT_SAFE_CALL(hipfftExecC2C(plan, d_FFTo_cc, d_SLM_cc, HIPFFT_BACKWARD));
					hipDeviceSynchronize();
					PropagateToSLMDC_Fresnel<<< n_blocks_Phi, BLOCK_SIZE >>>(d_obtainedPhase, d_weights, d_SLM_cc, d_pSLM_f, l, d_pSLMstart_f, (l==(N_iterations-1)), d_pSLM_uc);
				}
				else
				{
					PropagateToSLM_Fresnel<<< n_blocks_Phi, BLOCK_SIZE >>>(d_spotRe_f, d_spotIm_f, d_pSLM_f, d_weights, l, d_pSLMstart_f, d_Iobtained, (l==(N_iterations-1)), d_pSLM_uc, d_LUT_uc, d_AberrationCorr_f, d_LUTPolCoeff_f);
				}
				M_CHECK_ERROR();
				hipDeviceSynchronize();
			}
			/*cudaEventRecord(stop, 0);
			cudaEventSynchronize(stop);
			cudaEventElapsedTime(&deltaTime, start, stop);
			*gpuTime = deltaTime;	*/
			if (saveI_b)
				M_SAFE_CALL(hipMemcpy(h_Iobtained, d_Iobtained, N_spots*(N_iterations)*sizeof(float), hipMemcpyDeviceToHost));
			else
				M_SAFE_CALL(hipMemcpy(h_Iobtained, d_weights, N_spots*(N_iterations)*sizeof(float), hipMemcpyDeviceToHost));
			M_SAFE_CALL(hipMemcpy(h_pSLM_uc, d_pSLM_uc, memsize_SLMuc, hipMemcpyDeviceToHost));
			break;
		case 2:
			//generate hologram using fast fourier transforms
			//Uncomment this to start with pre-calculated hologram:
			//cudaMemcpy(d_pSLM_uc, h_pSLM_uc, memsize_SLMuc, cudaMemcpyHostToDevice);
			//cudaDeviceSynchronize();
			//p_uc2c_cc_shift<<< n_blocks_Phi, BLOCK_SIZE >>>(d_SLM_cc, d_pSLM_uc, N_pixels, data_w);
			//M_DISPLAY_DATA_CC(d_SLM_cc, 100);
			M_SAFE_CALL(hipMemcpy(d_desiredAmp, h_desiredAmp, memsize_spotsf, hipMemcpyHostToDevice));
			M_SAFE_CALL(hipMemset(d_FFTd_cc, 0, memsize_SLMcc));
			M_CHECK_ERROR();
			hipDeviceSynchronize();
			for (int l=0; l<N_iterations; l++)
			{

				// Transform to trapping plane
				M_CUFFT_SAFE_CALL(hipfftExecC2C(plan, d_SLM_cc, d_FFTo_cc, HIPFFT_FORWARD));
				hipDeviceSynchronize();

				// Copy phases for spot indices in d_FFTo_cc to d_FFTd_cc
				if (useDC_b)
					ReplaceAmpsSpotsDC_FFT <<< n_blocks_Phi, BLOCK_SIZE >>> (d_FFTo_cc, d_FFTd_cc, l, d_Iobtained, d_weights, (l==(N_iterations-1)));
				else
					ReplaceAmpsSpots_FFT <<< 1, N_spots >>> (d_FFTo_cc, d_FFTd_cc, l, d_Iobtained, d_weights, (l==(N_iterations-1)));
				M_CHECK_ERROR();
				hipDeviceSynchronize();

				//Transform back to SLM plane
				M_CUFFT_SAFE_CALL(hipfftExecC2C(plan, d_FFTd_cc, d_SLM_cc, HIPFFT_BACKWARD));
				hipDeviceSynchronize();
				//M_DISPLAY_DATA_CC(d_SLM_cc, 100);

				// Set amplitudes in d_SLM to the laser amplitude profile
				ReplaceAmpsSLM_FFT <<< n_blocks_Phi, BLOCK_SIZE >>> (d_aLaserFFT, d_SLM_cc, d_pSLMstart_f, (l==(N_iterations-1)), d_pSLM_uc, d_LUT_uc, d_AberrationCorr_f, d_LUTPolCoeff_f);
				M_CHECK_ERROR();
				//M_DISPLAY_DATA_CC(d_SLM_cc, 100);
				hipDeviceSynchronize();
			}
			if (saveI_b)
				M_SAFE_CALL(hipMemcpy(h_Iobtained, d_Iobtained, N_spots*(N_iterations)*sizeof(float), hipMemcpyDeviceToHost));
			else
				M_SAFE_CALL(hipMemcpy(h_Iobtained, d_weights, N_spots*(N_iterations)*sizeof(float), hipMemcpyDeviceToHost));
			M_SAFE_CALL(hipMemcpy(h_pSLM_uc, d_pSLM_uc, memsize_SLMuc, hipMemcpyDeviceToHost));
			break;
		default:
			break;
	}
	//Handle CUDA errors
	status = hipGetLastError();
	return status;
}

//Allocate GPU memory
extern "C" __declspec(dllexport)  int startCUDA(float *h_pSLMstart, int deviceId)
{
	//Make sure GPU with desired deviceId exists, set deviceId to 0 if not
	int deviceCount=0;
	if (hipGetDeviceCount(&deviceCount)!=0)
	if (deviceId>=deviceCount)
	{
		deviceId=0;
	}
	M_SAFE_CALL(hipSetDevice(deviceId));
	hipDeviceProp_t deviceProp;
    M_SAFE_CALL(hipGetDeviceProperties(&deviceProp, deviceId));
    maxThreads_device = deviceProp.maxThreadsPerBlock;
	borderWidthDC_i = 0;
	int MaxIterations = 1000;
	data_w = SLM_SIZE;
	hipMemcpyToSymbol(HIP_SYMBOL(c_data_w), &data_w, sizeof(int), 0, hipMemcpyHostToDevice);
	float data_w_f = (float)data_w;
	hipMemcpyToSymbol(HIP_SYMBOL(c_data_w_f), &data_w_f, sizeof(float), 0, hipMemcpyHostToDevice);
	int half_w = (int)(data_w/2);
	hipMemcpyToSymbol(HIP_SYMBOL(c_half_w), &half_w, sizeof(int), 0, hipMemcpyHostToDevice);
	float half_w_f = (float)data_w/2.0f;
	hipMemcpyToSymbol(HIP_SYMBOL(c_half_w_f), &half_w_f, sizeof(float), 0, hipMemcpyHostToDevice);
	N_pixels = data_w * data_w;
	hipMemcpyToSymbol(HIP_SYMBOL(c_N_pixels), &N_pixels, sizeof(int), 0, hipMemcpyHostToDevice);
	float N_pixels_f = (float)N_pixels;
	hipMemcpyToSymbol(HIP_SYMBOL(c_N_pixels_f), &N_pixels_f, sizeof(float), 0, hipMemcpyHostToDevice);
	int logN = (int)(log2(data_w_f));
	hipMemcpyToSymbol(HIP_SYMBOL(c_log2data_w), &logN, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_useRPC_b), &useRPC_b, sizeof(bool), 0, hipMemcpyHostToDevice);
	float SLMpitch_f = 1.0f/data_w_f;
	hipMemcpyToSymbol(HIP_SYMBOL(c_SLMpitch_f), &SLMpitch_f, sizeof(float), 0, hipMemcpyHostToDevice);
	N_iterations_last = 10;
	memsize_spotsf = MAX_SPOTS * sizeof(float);
	memsize_SLM_f = N_pixels * sizeof(float);
    memsize_SLMuc = N_pixels * sizeof(uint16_t);
	memsize_SLMcc = N_pixels * sizeof(hipfftComplex);
    n_blocks_Phi = (N_pixels/BLOCK_SIZE + (N_pixels%BLOCK_SIZE == 0 ? 0:1));

	//memory allocations for all methods
	M_SAFE_CALL(hipMalloc((void**)&d_x, memsize_spotsf ));
	M_SAFE_CALL(hipMalloc((void**)&d_y, memsize_spotsf ));
	M_SAFE_CALL(hipMalloc((void**)&d_z, memsize_spotsf ));
	M_SAFE_CALL(hipMalloc((void**)&d_I, memsize_spotsf ));
	M_SAFE_CALL(hipMalloc((void**)&d_desiredAmp, memsize_spotsf ));
	M_SAFE_CALL(hipMalloc((void**)&d_weights, MAX_SPOTS*(MaxIterations+1)*sizeof(float)));
	M_SAFE_CALL(hipMalloc((void**)&d_Iobtained, MAX_SPOTS*MaxIterations*sizeof(float)));
	M_SAFE_CALL(hipMalloc((void**)&d_obtainedPhase, memsize_spotsf ));
	M_SAFE_CALL(hipMalloc((void**)&d_spotRe_f, memsize_spotsf ));
	M_SAFE_CALL(hipMalloc((void**)&d_spotIm_f, memsize_spotsf ));
	int data_w_pow2 = pow(2, ceil(log((float)data_w)/log(2.0f)));
	M_SAFE_CALL(hipMalloc((void**)&d_pSLM_f, data_w_pow2*data_w_pow2*sizeof(float)));//the size of d_pSLM_f must be a power of 2 for the summation algorithm to work
	M_SAFE_CALL(hipMemset(d_pSLM_f, 0, data_w_pow2*data_w_pow2*sizeof(float)));
	M_SAFE_CALL(hipMalloc((void**)&d_pSLMstart_f, memsize_SLM_f));
	M_SAFE_CALL(hipMalloc((void**)&d_pSLM_uc, memsize_SLMuc));
	M_SAFE_CALL(hipMemset(d_pSLMstart_f, 0, N_pixels*sizeof(float)));
	M_SAFE_CALL(hipMemcpy(d_pSLM_f, h_pSLMstart, N_pixels*sizeof(float), hipMemcpyHostToDevice));

	//memory allocations etc. for all FFT based Gerchberg-Saxton
	M_SAFE_CALL(hipMalloc((void**)&d_spot_index, MAX_SPOTS * sizeof(int)));
	M_SAFE_CALL(hipMalloc((void**)&d_FFTd_cc, memsize_SLMcc));
	M_SAFE_CALL(hipMalloc((void**)&d_FFTo_cc, memsize_SLMcc));
	M_SAFE_CALL(hipMalloc((void**)&d_SLM_cc, memsize_SLMcc));
	M_SAFE_CALL(hipDeviceSynchronize());
	p2c <<< n_blocks_Phi, BLOCK_SIZE >>>(d_SLM_cc, d_pSLM_f, N_pixels);
	M_CHECK_ERROR();
	hipDeviceSynchronize();
	M_CUFFT_SAFE_CALL(hipfftPlan2d(&plan, data_w, data_w, HIPFFT_C2C));
	float *h_aLaserFFT = (float *)malloc(memsize_SLM_f);

	status = hipGetLastError();
	return status;
}

//Free GPU memory and shut down SLM
extern "C" __declspec(dllexport)  int stopCUDA()
{
	M_SAFE_CALL(hipFree(d_x));
	M_SAFE_CALL(hipFree(d_y));
	M_SAFE_CALL(hipFree(d_z));
	M_SAFE_CALL(hipFree(d_I));
	M_SAFE_CALL(hipFree(d_weights));
	M_SAFE_CALL(hipFree(d_Iobtained));
	M_SAFE_CALL(hipFree(d_pSLM_f));
	M_SAFE_CALL(hipFree(d_pSLMstart_f));
	M_SAFE_CALL(hipFree(d_pSLM_uc));
	M_SAFE_CALL(hipFree(d_FFTd_cc));
	M_SAFE_CALL(hipFree(d_FFTo_cc));
	M_SAFE_CALL(hipFree(d_SLM_cc));
	M_CUFFT_SAFE_CALL(hipfftDestroy(plan));
	hipDeviceReset();

	status = hipGetLastError();
	return status;
}

//Device functions
__device__ float uc2phase(float uc)
{
	return (float)uc*2.0f*M_PI/65536.0f - M_PI;
}
__device__ uint16_t phase2uc(float phase2pi)
{
	return (uint16_t)floor((phase2pi + M_PI)*65536.0f / (2.0f * M_PI));
}
__device__ int phase2int32(float phase2pi)
{
	return (int)floor((phase2pi + M_PI)*65536.0f / (2.0f * M_PI));
}
__device__ float ApplyAberrationCorrection(float pSpot, float correction)
{
		pSpot = pSpot - correction;		//apply correction
		return (pSpot - (2.0f*M_PI) * floor((pSpot+M_PI) / (2.0f*M_PI))); //apply mod([-pi, pi], pSpot)
}
__device__ int getXint(int index)
{
#ifdef SLMPOW2
	int X_int = index&(c_data_w[0]-1);
#else
	float X_int= index%c_data_w[0];
#endif
	return X_int;
}
__device__ int getYint(int index, int X_int)
{
#ifdef SLMPOW2
 	int Y_int = (index-X_int)>>c_log2data_w[0];
#else
	int Y_int = (float)(floor((float)index/c_data_w_f[0]));
#endif
	return Y_int;
}
__device__ int fftshift(int idx, int X, int Y)
{
	if (X < c_half_w[0])
	{
		if (Y < c_half_w[0])
		{
			return idx + (c_data_w[0] * c_half_w[0]) + c_half_w[0];
		}
		else
		{
			return idx - (c_data_w[0] * c_half_w[0]) + c_half_w[0];
		}
	}
	else
	{
		if (Y < c_half_w[0])
		{
			return idx + (c_data_w[0] * c_half_w[0]) - c_half_w[0];
		}
		else
		{
			return idx - (c_data_w[0] * c_half_w[0]) - c_half_w[0];
		}
	}
}
__device__ void warpReduceC(volatile float *s_Vre, volatile float *s_Vim, int tid)
{
	s_Vre[tid] += s_Vre[tid + 32];
	s_Vim[tid] += s_Vim[tid + 32];
	s_Vre[tid] += s_Vre[tid + 16];
	s_Vim[tid] += s_Vim[tid + 16];
	s_Vre[tid] += s_Vre[tid + 8];
	s_Vim[tid] += s_Vim[tid + 8];
	s_Vre[tid] += s_Vre[tid + 4];
	s_Vim[tid] += s_Vim[tid + 4];
	s_Vre[tid] += s_Vre[tid + 2];
	s_Vim[tid] += s_Vim[tid + 2];
	s_Vre[tid] += s_Vre[tid + 1];
	s_Vim[tid] += s_Vim[tid + 1];
}
inline int computeAndCopySpotData(float *h_I, float *x, float *y, float *z, int N_spots, int method)
{
	//float Isum = 0.0f;
	//for (int i = 0; i<N_spots; i++)
	//	Isum += h_I[i];
	for (int j = 0; j<N_spots; j++)
	{
		float sincx_rec = (x[j]==0)? 1.0f:((M_PI*x[j]/SLMsizef)/sinf(M_PI*x[j]/SLMsizef));
		float sincy_rec = (y[j]==0)? 1.0f:((M_PI*y[j]/SLMsizef)/sinf(M_PI*y[j]/SLMsizef));
		h_desiredAmp[j] = (h_I[j] <= 0.0f) ? 1.0f:(sincx_rec * sincy_rec * sqrtf(h_I[j]/100)*SLMsizef*SLMsizef);
		if (method == 2)
			h_spotIndex[j] = ((int)(x[j])&(data_w-1))  + ((int)(y[j])&(data_w-1))* data_w;
	}
	hipMemcpyToSymbol(HIP_SYMBOL(c_x), x, N_spots*sizeof(float), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_y), y, N_spots*sizeof(float), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_z), z, N_spots*sizeof(float), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_desiredAmp), h_desiredAmp, N_spots*sizeof(float), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_N_spots), &N_spots, sizeof(int), 0, hipMemcpyHostToDevice);
	if (method == 2)
		hipMemcpyToSymbol(HIP_SYMBOL(c_spotIndex), h_spotIndex, N_spots*sizeof(int), 0, hipMemcpyHostToDevice);
	if (N_spots == 0)
		method = -1;
	return method;
}

//Apply corrections to precalculated hologram
__global__ void ApplyCorrections(uint16_t *g_pSLM_uc, uint16_t *g_LUT, float *g_AberrationCorr_f, float *g_LUTPolCoeff_f)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float pSLM2pi_f = uc2phase(g_pSLM_uc[idx]);
	g_pSLM_uc[idx] = phase2uc(pSLM2pi_f);
}
//Calculate hologram using "Lenses and Prisms"
__global__ void LensesAndPrisms(uint16_t *g_SLMuc, uint16_t *g_LUT, float *d_AberrationCorr_f, float *d_LUTPolCoeff_f)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < c_N_pixels[0])
	{
		//get pixel coordinates
		int X_int = getXint(idx);
		int Y_int = getYint(idx, X_int);
		float X = c_SLMpitch_f[0]*(X_int - c_half_w_f[0]);
		float Y = c_SLMpitch_f[0]*(Y_int - c_half_w_f[0]);
		float phase2pi;
		float SLMre = 0.0f;
		float SLMim = 0.0f;
		for (int ii=0; ii<c_N_spots[0]; ++ii)
		{
			//add variable phases to function call
			phase2pi = M_PI * c_z[ii] * (X*X + Y*Y) + 2.0f * M_PI * (X * (c_x[ii]) + Y * (c_y[ii]) );
			SLMre = SLMre + c_desiredAmp[ii] * cosf(phase2pi);
			SLMim = SLMim + c_desiredAmp[ii] * sinf(phase2pi);
		}
		phase2pi = atan2f(SLMim, SLMre);	// [-pi,pi]
		g_SLMuc[idx] = phase2uc(phase2pi);
	}
}
__global__ void calculateIobtained(uint16_t *g_pSLM_uc, float *g_Iobtained)
{
	int blockSize = c_data_w[0];
	int spot_number = blockIdx.x;
	int tid = threadIdx.x;
	int i = tid;
	__shared__ float s_Vre[SLM_SIZE];
	__shared__ float s_Vim[SLM_SIZE];
	s_Vre[tid] = 0.0f;
	s_Vim[tid] = 0.0f;
	float X = c_SLMpitch_f[0] * ((float)tid - c_half_w_f[0]);
	float Y = - c_SLMpitch_f[0] * c_half_w_f[0];
	float pSLM_1;
	float p;
	while (i < c_N_pixels[0])
	{
		pSLM_1 = 2.0f*M_PI*(float)g_pSLM_uc[i]/65535.0f - M_PI;
		p = pSLM_1 - M_PI * (c_z[spot_number] * (X*X + Y*Y) + 2 * (X * c_x[spot_number] + Y * c_y[spot_number]));
		s_Vre[tid] += cosf(p);
		s_Vim[tid] += sinf(p);
		i += blockSize;
		Y += c_SLMpitch_f[0];
	}
	/*__syncthreads();
	if (tid < 512)
	{
		s_Vre[tid] += s_Vre[tid + 512];
		s_Vim[tid] += s_Vim[tid + 512];
	} */
	__syncthreads();
	if ((tid < 256)&&(SLM_SIZE>256))
	{
		s_Vre[tid] += s_Vre[tid + 256];
		s_Vim[tid] += s_Vim[tid + 256];
	}
	__syncthreads();
	if (tid < 128)
	{
		s_Vre[tid] += s_Vre[tid + 128];
		s_Vim[tid] += s_Vim[tid + 128];
	}
	__syncthreads();
	if (tid < 64)
	{
		s_Vre[tid] += s_Vre[tid + 64];
		s_Vim[tid] += s_Vim[tid + 64];
	}
	__syncthreads();
	if (tid < 32)
		warpReduceC(s_Vre, s_Vim, tid);
	if (tid == 0)
	{
		float spotRe_f = s_Vre[0] / c_N_pixels_f[0];			//512!
		float spotIm_f = s_Vim[0] / c_N_pixels_f[0];
		float amp = hypotf(spotRe_f, spotIm_f);
		g_Iobtained[spot_number] = amp*amp;
	}
}
__global__ void calculateIandPhase(uint16_t *g_pSLM_uc, float *g_Iobtained, float *g_Pobtained)
{
	int blockSize = c_data_w[0];
	int spot_number = blockIdx.x;
	int tid = threadIdx.x;
	int i = tid;
	__shared__ float s_Vre[SLM_SIZE];
	__shared__ float s_Vim[SLM_SIZE];
	s_Vre[tid] = 0.0f;
	s_Vim[tid] = 0.0f;
	float X = c_SLMpitch_f[0] * ((float)tid - c_half_w_f[0]);
	float Y = - c_SLMpitch_f[0] * c_half_w_f[0];
	float pSLM_1;
	float p;
	while (i < c_N_pixels[0])
	{
		pSLM_1 = 2.0f*M_PI*(float)g_pSLM_uc[i]/65535.0f - M_PI;
		p = pSLM_1 - M_PI * (c_z[spot_number] * (X*X + Y*Y) + 2 * (X * c_x[spot_number] + Y * c_y[spot_number]));
		s_Vre[tid] += cosf(p+2*M_PI*c_z[spot_number]);
		s_Vim[tid] += sinf(p+2*M_PI*c_z[spot_number]);
		i += blockSize;
		Y += c_SLMpitch_f[0];
	}
	/*__syncthreads();
	if (tid < 512)
	{
		s_Vre[tid] += s_Vre[tid + 512];
		s_Vim[tid] += s_Vim[tid + 512];
	} */
	__syncthreads();
	if ((tid < 256)&&(SLM_SIZE>256))
	{
		s_Vre[tid] += s_Vre[tid + 256];
		s_Vim[tid] += s_Vim[tid + 256];
	}
	__syncthreads();
	if (tid < 128)
	{
		s_Vre[tid] += s_Vre[tid + 128];
		s_Vim[tid] += s_Vim[tid + 128];
	}
	__syncthreads();
	if (tid < 64)
	{
		s_Vre[tid] += s_Vre[tid + 64];
		s_Vim[tid] += s_Vim[tid + 64];
	}
	__syncthreads();
	if (tid < 32)
		warpReduceC(s_Vre, s_Vim, tid);
	if (tid == 0)
	{
		float spotRe_f = s_Vre[0] / c_N_pixels_f[0];			//512!
		float spotIm_f = s_Vim[0] / c_N_pixels_f[0];
		float amp = hypotf(spotRe_f, spotIm_f);
		g_Pobtained[spot_number] = atan2f(spotIm_f , spotRe_f);
		g_Iobtained[spot_number] = amp*amp;
	}
}

//Functions for GS with Fresnel propagation

//Propagate from the SLM to the spot positions using Fresnel summation
//works only for blocksize = SLMsize
__global__ void PropagateToSpotPositions_Fresnel(float *g_pSLM2pi, float *g_spotRe_f, float *g_spotIm_f)
{
	int spot_number = blockIdx.x;
	int tid = threadIdx.x;
	int i = tid;
	__shared__ float s_Vre[SLM_SIZE];
	__shared__ float s_Vim[SLM_SIZE];
	s_Vre[tid] = 0.0f;
	s_Vim[tid] = 0.0f;
	int blockSize = blockDim.x;
	float X = c_SLMpitch_f[0] * ((float)tid - c_half_w_f[0]);
	float Y = - c_SLMpitch_f[0] * c_half_w_f[0];
	float p;
	while (i < c_N_pixels[0])
	{
		p = g_pSLM2pi[i] - M_PI * (c_z[spot_number] * (X*X + Y*Y) + 2.0f * (X * c_x[spot_number] + Y * c_y[spot_number]));
		s_Vre[tid] += cosf(p);
		s_Vim[tid] += sinf(p);
		i += blockSize;
		Y += c_SLMpitch_f[0];
	}
	/*__syncthreads();
	if (tid < 512)
	{
		s_Vre[tid] += s_Vre[tid + 512];
		s_Vim[tid] += s_Vim[tid + 512];
	} */
	__syncthreads();
	if ((tid < 256)&&(SLM_SIZE>256))
	{
		s_Vre[tid] += s_Vre[tid + 256];
		s_Vim[tid] += s_Vim[tid + 256];
	}
	__syncthreads();
	if (tid < 128)
	{
		s_Vre[tid] += s_Vre[tid + 128];
		s_Vim[tid] += s_Vim[tid + 128];
	}
	__syncthreads();
	if (tid < 64)
	{
		s_Vre[tid] += s_Vre[tid + 64];
		s_Vim[tid] += s_Vim[tid + 64];
	}
	__syncthreads();
	if (tid < 32)
		warpReduceC(s_Vre, s_Vim, tid);
	if (tid == 0)
	{
		g_spotRe_f[spot_number] = s_Vre[0];// / c_N_pixels_f[0];
		g_spotIm_f[spot_number] = s_Vim[0];// / c_N_pixels_f[0];
	}
}
//Propagate from the SLM to the spot positions using Fresnel summation
//works only for blocksize = SLMsize
__global__ void PropagateToSpotPositionsDC_Fresnel(float *g_pSLM_f, float *g_obtainedPhase, float *g_weights, float *obtainedI, int iteration)
{
	int spot_number = blockIdx.x;
	int tid = threadIdx.x;
	int i = tid;
	__shared__ float s_Vre[SLM_SIZE];
	__shared__ float s_Vim[SLM_SIZE];
	float X, Y;
	float p;
	s_Vre[tid] = 0.0f;
	s_Vim[tid] = 0.0f;
	int X_int = getXint(i);
	X = c_SLMpitch_f[0]*(X_int - c_half_w_f[0]);
	Y = -0.5f;
	while (i < c_N_pixels[0])
	{
		p = g_pSLM_f[i] - M_PI * (c_z[spot_number] * (X*X + Y*Y) + 2.0f * (X * c_x[spot_number] + Y * c_y[spot_number]));
		s_Vre[tid] += cosf(p);
		s_Vim[tid] += sinf(p);
		Y += c_SLMpitch_f[0];
		i += SLM_SIZE;
	}
	__syncthreads();
	if ((tid < 256)&&(SLM_SIZE>256))
	{
		s_Vre[tid] += s_Vre[tid + 256];
		s_Vim[tid] += s_Vim[tid + 256];
	}
	__syncthreads();
	if (tid < 128)
	{
		s_Vre[tid] += s_Vre[tid + 128];
		s_Vim[tid] += s_Vim[tid + 128];
	}
	__syncthreads();
	if (tid < 64)
	{
		s_Vre[tid] += s_Vre[tid + 64];
		s_Vim[tid] += s_Vim[tid + 64];
	}
	__syncthreads();
	if (tid < 32)
		warpReduceC(s_Vre, s_Vim, tid);
	if (tid == 0)
	{
		g_obtainedPhase[spot_number] = atan2f(s_Vim[0], s_Vre[0]);
		float obtainedAmp = hypotf(s_Vre[0], s_Vim[0]);
		float desiredAmp = c_desiredAmp[spot_number];
		if (iteration != 0)
		{
			g_weights[spot_number + c_N_spots[0]*iteration] = g_weights[spot_number + c_N_spots[0]*(iteration-1)] * (desiredAmp / obtainedAmp);
		}
		else
		{
			//obtainedAmp = (obtainedAmp<0.5f) ? 0.5f : obtainedAmp;
			g_weights[spot_number] = desiredAmp/c_N_pixels_f[0];
		}
		if (c_saveI_b[0])
			obtainedI[spot_number + c_N_spots[0]*iteration] = obtainedAmp*obtainedAmp/(desiredAmp*desiredAmp);//(c_N_pixels_f[0]*c_N_pixels_f[0]);
	}
}
//Obtain phases in SLM plane
__global__ void PropagateToSLM_Fresnel(float *g_spotRe_f, float *g_spotIm_f, float *g_pSLM2pi, float *g_weights, int iteration, float *g_pSLMstart, float *g_Iobtained, bool getpSLM65535, uint16_t *g_pSLM65535_uc,
								uint16_t *g_LUT, float *g_AberrationCorr_f, float *g_LUTPolCoeff_f)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	__shared__ float s_aSpot[MAX_SPOTS], s_aSpotsMean, s_weight[MAX_SPOTS], s_pSpot[MAX_SPOTS];
	float reSLM = 0.0f, imSLM = 0.0f, pSLM2pi_f = 0.0f;
	if (idx<c_N_pixels[0])
	{
		if (tid<c_N_spots[0])
		{
			float spotRe_f = g_spotRe_f[tid];
			float spotIm_f = g_spotIm_f[tid];
			s_pSpot[tid] = atan2f(spotIm_f, spotRe_f);
			s_aSpot[tid] = hypotf(spotRe_f, spotIm_f)/c_desiredAmp[tid];
			if (iteration != 0)
				s_weight[tid] = g_weights[tid + iteration*c_N_spots[0]];
			else
			{
				s_aSpot[tid] = (s_aSpot[tid]<0.5f) ? 0.5f : s_aSpot[tid];
				s_weight[tid] = c_desiredAmp[tid];
			}
		}
		__syncthreads();
		//compute weights
		if  (tid==0)
		{
			float s_aSpot_sum = 0.0f;
			for (int jj=0; jj<c_N_spots[0];jj++)
			{
				s_aSpot_sum += s_aSpot[jj];
			}
			s_aSpotsMean = s_aSpot_sum / (float)c_N_spots[0];
		}
		__syncthreads();
		if (tid<c_N_spots[0])
		{
			s_weight[tid] = s_weight[tid] * s_aSpotsMean / s_aSpot[tid];
			if (!getpSLM65535)											//Copy weights to use as initial value next run
				g_weights[tid + c_N_spots[0]*(iteration+1)] = s_weight[tid];
			//else
			//	g_weights[tid] = s_weight[tid];							//Transferring weights to next run may give diverging weights
			if (c_saveI_b[0])
				g_Iobtained[tid + c_N_spots[0]*iteration] = s_aSpot[tid]*s_aSpot[tid];			//may be excluded, used for monitoring only
		}
		__syncthreads();
		//get pixel coordinates
		int X_int = getXint(idx);
		int Y_int = getYint(idx, X_int);
		float X = c_SLMpitch_f[0]*(X_int - c_half_w_f[0]);
		float Y = c_SLMpitch_f[0]*(Y_int - c_half_w_f[0]);
		//compute SLM pSpot by summing contribution from all spots
		for (int k=0; k<c_N_spots[0]; k++)
		{
			float delta = M_PI * c_z[k] * (X*X + Y*Y) + 2.0f * M_PI * (X * c_x[k] + Y * c_y[k]);
			reSLM += s_weight[k] * cosf(s_pSpot[k] + delta);
			imSLM += s_weight[k] * sinf(s_pSpot[k] + delta);
		}
		pSLM2pi_f = atan2f(imSLM, reSLM);
		if (c_useRPC_b[0])			//Apply RPC (restricted Phase Change)
		{
			float pSLMstart = g_pSLMstart[idx];
			if (fabs(pSLM2pi_f - pSLMstart) > c_alphaRPC_f[0])
				pSLM2pi_f = pSLMstart;
			if (getpSLM65535)
				g_pSLMstart[idx] = pSLM2pi_f;
		}
		if (getpSLM65535)					//Compute final SLM phases and write to global memory...
			g_pSLM65535_uc[idx] = phase2uc(pSLM2pi_f);
		g_pSLM2pi[idx] = pSLM2pi_f;	//...or write intermediate pSpot to global memory
	}
}
//Obtain phases in SLM plane
__global__ void PropagateToSLMDC_Fresnel(float *g_pSpot, float *g_wSpot, hipfftComplex *g_cSLM_cc, float *g_pSLM_f, int iteration, float *g_pSLMstart, bool getpSLM65535,
										 uint16_t *g_pSLM65535_uc)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	__shared__ float s_weight[MAX_SPOTS], s_pSpot[MAX_SPOTS];
	float reSLM = 0.0f, imSLM = 0.0f, pSLM2pi_f = 0.0f;
	if (idx<c_N_pixels[0])
	{
		if (tid<c_N_spots[0])
		{
			s_pSpot[tid] = g_pSpot[tid];
			s_weight[tid] = g_wSpot[tid+c_N_spots[0]*iteration];
		}
		__syncthreads();
		//get pixel coordinates
		int X_int = getXint(idx);
		int Y_int = getYint(idx, X_int);
		int shiftedidx = fftshift(idx, X_int, Y_int);
		float X = c_SLMpitch_f[0]*(X_int - c_half_w_f[0]);
		float Y = c_SLMpitch_f[0]*(Y_int - c_half_w_f[0]);
		//compute SLM pSpot by summing contribution from all spots
		for (int k=0; k<c_N_spots[0]; k++)
		{
			float delta = M_PI * c_z[k] * (X*X + Y*Y) + 2.0f * M_PI * (X * c_x[k] + Y * c_y[k]);
			reSLM += s_weight[k] * cosf(s_pSpot[k] + delta);
			imSLM += s_weight[k] * sinf(s_pSpot[k] + delta);
		}
		hipfftComplex cSLM_cc = g_cSLM_cc[shiftedidx];
		reSLM += cSLM_cc.x/c_N_pixels_f[0];
		imSLM += cSLM_cc.y/c_N_pixels_f[0];
		pSLM2pi_f = atan2f(imSLM, reSLM);
		if (c_useRPC_b[0])			//Apply RPC (restricted Phase Change)
		{
			float pSLMstart = g_pSLMstart[shiftedidx];
			if (fabs(pSLM2pi_f - pSLMstart) > c_alphaRPC_f[0])
				pSLM2pi_f = pSLMstart;
			if (getpSLM65535)
				g_pSLMstart[shiftedidx] = pSLM2pi_f;
		}
		g_pSLM_f[idx] = pSLM2pi_f;
		g_cSLM_cc[shiftedidx].x = cosf(pSLM2pi_f);
		g_cSLM_cc[shiftedidx].y = sinf(pSLM2pi_f);
		if (getpSLM65535)					//Compute final SLM phases and write to global memory...
			g_pSLM65535_uc[idx] = phase2uc(pSLM2pi_f);
	}
}
//Clear inside the DC frame
__global__ void setActiveRegionToZero(hipfftComplex *g_Farfield_cc) //this only works if blocksize = nblocks = SLMsize = 512
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int idx = bid * blockDim.x + tid;
	if (((tid < (c_half_w[0] - c_DCborderWidth[0]))||(tid > ((c_half_w[0]-1) + c_DCborderWidth[0])))&&((bid < (c_half_w[0] - c_DCborderWidth[0]))||(bid > ((c_half_w[0]-1) + c_DCborderWidth[0]))))
	{
		g_Farfield_cc[idx].x = 0.0f;
		g_Farfield_cc[idx].y = 0.0f;
	}
}

//Functions for GS with FFT propagation

//Compute the phase in SLM pixels and set amplitude to unity or Laser amp
__global__ void ReplaceAmpsSLM_FFT(float *g_aLaser, hipfftComplex *g_cAmp, float *g_pSLMstart, bool getpSLM65535, uint16_t *g_pSLM65535_uc,
								uint16_t *g_LUT, float *g_AberrationCorr_f, float *g_LUTPolCoeff_f)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<c_N_pixels[0])
	{
		float aLaser = 1.0f;//g_aLaser[idx];
		hipfftComplex cAmp = g_cAmp[idx];
 		float pSLM2pi_f = atan2f(cAmp.y, cAmp.x);
 		if (c_useRPC_b[0])
		{
			float pSLMstart = g_pSLMstart[idx];
			if (fabs(pSLM2pi_f - pSLMstart) > c_alphaRPC_f[0])
				pSLM2pi_f = pSLMstart;
		}
		if (getpSLM65535)
		{
			if (c_useRPC_b[0])
				g_pSLMstart[idx] = pSLM2pi_f;
			//float phase65535;
			int X_int = getXint(idx);
			int Y_int = getYint(idx, X_int);
			int shiftedidx = fftshift(idx, X_int, Y_int);
			g_pSLM65535_uc[shiftedidx] = phase2uc(pSLM2pi_f);
		}
		g_cAmp[idx].x = aLaser*cosf(pSLM2pi_f);
		g_cAmp[idx].y = aLaser*sinf(pSLM2pi_f);
	}
	__syncthreads();
}

//Adjust amplitudes in spot positions
__global__ void ReplaceAmpsSpots_FFT(hipfftComplex *g_cSpotAmp_cc, hipfftComplex *g_cSpotAmpNew_cc, int iteration, float *g_Iobtained, float *g_weight, bool last_iteration)
{
	int tid = threadIdx.x;
	int spotIndex;
	float pSpot;
	__shared__ float s_aSpot[MAX_SPOTS], s_ISpotsMeanSq;
	float weight;
	hipfftComplex cSpotAmp_cc;
	if (tid<c_N_spots[0])
	{
		spotIndex = c_spotIndex[tid];
		cSpotAmp_cc = g_cSpotAmp_cc[spotIndex];
		pSpot = atan2f(cSpotAmp_cc.y, cSpotAmp_cc.x);
		s_aSpot[tid] = hypotf(cSpotAmp_cc.x, cSpotAmp_cc.y)/c_desiredAmp[tid];
		if (iteration != 0)
			weight = g_weight[tid + iteration*c_N_spots[0]];
		else
		{
			s_aSpot[tid] = (s_aSpot[tid]<0.5f) ? 0.5f : s_aSpot[tid];
			weight = c_desiredAmp[tid];
		}
	}
	__syncthreads();
	//compute weights
	if  (tid==0)
	{
		float ISpot_sum = 0.0f;
		for (int jj=0; jj<c_N_spots[0];jj++)
		{
			ISpot_sum += s_aSpot[jj]*s_aSpot[jj];
		}
		s_ISpotsMeanSq = sqrtf(ISpot_sum / (float)c_N_spots[0]);				//integer division!!
	}
	__syncthreads();
	if (tid<c_N_spots[0])
	{
		weight = weight * s_ISpotsMeanSq / s_aSpot[tid];
		cSpotAmp_cc.x = cosf(pSpot) * weight;
		cSpotAmp_cc.y = sinf(pSpot) * weight;
		g_cSpotAmpNew_cc[spotIndex] = cSpotAmp_cc;
		if (last_iteration)
			g_weight[tid] = weight;
		else
			g_weight[c_N_spots[0] * (iteration + 1) + tid] = weight;
		if (c_saveI_b[0])
			g_Iobtained[c_N_spots[0] * (iteration) + tid] = s_aSpot[tid]*s_aSpot[tid];
	}
}

//Adjust amplitudes in spot positions
__global__ void ReplaceAmpsSpotsDC_FFT(hipfftComplex *g_cSpotAmp_cc, hipfftComplex *g_cSpotAmpNew_cc, int iteration, float *g_Iobtained, float *g_weight, bool last_iteration)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int spotIndex;
	float pSpot;
	//__shared__ float s_aSpot[MAX_SPOTS], s_ISpotsMeanSq;
	float weight;
	hipfftComplex cSpotAmp_cc;
	if (idx<c_N_spots[0])
	{
		spotIndex = c_spotIndex[idx];
		cSpotAmp_cc = g_cSpotAmp_cc[spotIndex];
		pSpot = atan2f(cSpotAmp_cc.y, cSpotAmp_cc.x);
		float aSpot = hypotf(cSpotAmp_cc.x, cSpotAmp_cc.y)/c_desiredAmp[idx];
		if (iteration != 0)
			weight = g_weight[idx + iteration*c_N_spots[0]];
		else
		{
			aSpot = (aSpot<0.5f) ? 0.5f : aSpot; //ska det vara s� h�r med DC?
			weight = c_desiredAmp[idx]/(c_N_pixels_f[0]);
		}
		weight = weight / aSpot;
		cSpotAmp_cc.x = cosf(pSpot) * weight;
		cSpotAmp_cc.y = sinf(pSpot) * weight;
		g_cSpotAmpNew_cc[spotIndex] = cSpotAmp_cc;
		if (last_iteration)
			g_weight[idx] = weight;
		else
			g_weight[c_N_spots[0] * (iteration + 1) + idx] = weight;
		if (c_saveI_b[0])
			g_Iobtained[c_N_spots[0] * (iteration) + idx] = aSpot*aSpot;
	}
	int X_int = getXint(idx);
	int Y_int = getYint(idx, X_int);
	if (((X_int > (c_half_w[0] - c_DCborderWidth[0]))&&(X_int < ((c_half_w[0]-1) + c_DCborderWidth[0])))||((Y_int > (c_half_w[0] - c_DCborderWidth[0]))&&(Y_int < ((c_half_w[0]-1) + c_DCborderWidth[0]))))
	{
		g_cSpotAmpNew_cc[idx].x = g_cSpotAmp_cc[idx].x/(c_N_pixels_f[0]);
		g_cSpotAmpNew_cc[idx].y = g_cSpotAmp_cc[idx].y/(c_N_pixels_f[0]);
	}
}

//Misc help functions

__global__ void testfunc(float *testdata)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	testdata[idx] = idx;
}
//Convert from uint16_t [0, 65535] to float [-pi, pi]
__global__ void uc2f(float *f, uint16_t *uc, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<N)
	{
		f[idx] = uc[idx]*2.0f*M_PI/65536.0f - M_PI;
	}
}
//Calculate complex from phases
__global__ void p2c(hipfftComplex *g_c, float *g_p, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<M)
	{
		float pSpot = g_p[idx];
 		g_c[idx].x = cosf(pSpot);
		g_c[idx].y = sinf(pSpot);
	}
	__syncthreads();
}

//Calculate amplitudes from complex
__global__ void c_cc2a_f(float *g_a, hipfftComplex *g_c, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<M)
	{
		g_a[idx] = hypotf(g_c[idx].x, g_c[idx].y);
	}
	__syncthreads();
}
//Calculate phases from complex
__global__ void c_cc2p_cc(hipfftComplex *g_p, hipfftComplex *g_c, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<M)
	{
		g_p[idx].x = atan2f(g_c[idx].y, g_c[idx].x);
		g_p[idx].y = 0.0f;
	}
	__syncthreads();
}
//Calculate phases from complex
__global__ void c_cc2p_f(float *g_p, hipfftComplex *g_c, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<M)
	{
		g_p[idx] = atan2f(g_c[idx].y, g_c[idx].x);
	}
	__syncthreads();
}

//Copy real part from complex
__global__ void c_cc2re_f(float *g_p, hipfftComplex *g_c, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<M)
	{
		g_p[idx] = g_c[idx].x;
	}
	__syncthreads();
}
//Copy imaginary part from complex
__global__ void c_cc2im_f(float *g_p, hipfftComplex *g_c, int M)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<M)
	{
		g_p[idx] = g_c[idx].y;
	}
	__syncthreads();
}

//Custom debug functions

inline void mSafeCall(hipError_t status, int line, const char *file)
{
#ifdef M_CUDA_DEBUG
	do
	{
		if(status != cudaSuccess)
		{
			char CUDAmessage[200] = "CUDA says: ";
			strcat(CUDAmessage,	cudaGetErrorString(status));
			sprintf(CUDAmessage,	"%s\non line: %d\n", CUDAmessage, line);
			//AfxMessageBox(CUDAmessage);
			if (status != CUFFT_SUCCESS)
			exit(-1);
		}
		cudaDeviceSynchronize();
		status = cudaGetLastError();
		if(status!=cudaSuccess)
		{
			char CUDAmessage[200] = "CUDA failed after sychronization:\n";
			strcat(CUDAmessage,	cudaGetErrorString(status));
			sprintf(CUDAmessage,	"%s\non line: %d\n", CUDAmessage, line);
			//AfxMessageBox(CUDAmessage);
			exit(-1);
		}
	}while(0);
#endif
	return;
}
inline void mCufftSafeCall(hipfftResult_t status, int line, const char *file)
{
#ifdef M_CUDA_DEBUG
	if(status != CUFFT_SUCCESS)
	{
		char CUDAmessage[200] = "CUFFT error, CUDA says:\n ";
		switch (status) {
            case CUFFT_INVALID_PLAN:   strcat(CUDAmessage,"CUFFT_INVALID_PLAN\n");break;
            case CUFFT_ALLOC_FAILED:   strcat(CUDAmessage,"CUFFT_ALLOC_FAILED\n");break;
            case CUFFT_INVALID_TYPE:   strcat(CUDAmessage,"CUFFT_INVALID_TYPE\n");break;
            case CUFFT_INVALID_VALUE:  strcat(CUDAmessage,"CUFFT_INVALID_VALUE\n");break;
            case CUFFT_INTERNAL_ERROR: strcat(CUDAmessage,"CUFFT_INTERNAL_ERROR\n");break;
            case CUFFT_EXEC_FAILED:    strcat(CUDAmessage,"CUFFT_EXEC_FAILED\n");break;
            case CUFFT_SETUP_FAILED:   strcat(CUDAmessage,"CUFFT_SETUP_FAILED\n");break;
            case CUFFT_INVALID_SIZE:   strcat(CUDAmessage,"CUFFT_INVALID_SIZE\n");break;
            case CUFFT_UNALIGNED_DATA: strcat(CUDAmessage,"CUFFT_UNALIGNED_DATA\n");break;
            default: strcat(CUDAmessage,"CUFFT Unknown error code\n");
		}
		sprintf(CUDAmessage,	"%son line: %d\nin file: %s", CUDAmessage, line, file);
		//AfxMessageBox(CUDAmessage);
		exit(-1);
	}
	cudaDeviceSynchronize();
	cudaError_t status2 = cudaGetLastError();
	if(status2!=cudaSuccess)
	{
			char CUDAmessage[200] = "CUDA failed after sychronization:\n";
			strcat(CUDAmessage,	cudaGetErrorString(status2));
			sprintf(CUDAmessage,	"%s\non line: %d\n", CUDAmessage, line);
			//AfxMessageBox(CUDAmessage);
			exit(-1);
	}
#endif
	return;
}
inline void mCheckError(int line, const char *file)
{
#ifdef M_CUDA_DEBUG
	do
	{
		cudaError_t status = cudaGetLastError();
		if(status!=cudaSuccess)
		{
			char CUDAmessage[200] = "CUDA says: ";
			strcat(CUDAmessage,	cudaGetErrorString(status));
			sprintf(CUDAmessage,	"%s\non line: %d\n", CUDAmessage, line);
			//AfxMessageBox(CUDAmessage);
			exit(-1);
		}
		cudaDeviceSynchronize();
		status = cudaGetLastError();
		if(status!=cudaSuccess)
		{
			char CUDAmessage[200] = "CUDA failed after sychronization:\n";
			strcat(CUDAmessage,	cudaGetErrorString(status));
			sprintf(CUDAmessage,	"%s\non line: %d\n", CUDAmessage, line);
			//AfxMessageBox(CUDAmessage);
			exit(-1);
		}
	}while(0);
#endif
	return;
}

inline void mDisplayDataF(float *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
	do
	{
		int maxlength = 50;
		float *h_data;
		length = (length<=maxlength) ? length : maxlength;
		char MessageString[1000];
		h_data = (float*)malloc(length * sizeof (float));
		M_SAFE_CALL(cudaMemcpy(h_data, d_data, length*sizeof(float), cudaMemcpyDeviceToHost));
		sprintf(MessageString,	"Line: %d\nData: ", line);
		for (int ii = 0;ii<length;++ii)
		{
			sprintf(MessageString,	"%s %f", MessageString, h_data[ii]);
		}
		//AfxMessageBox(MessageString, MB_ICONINFORMATION);
		free(h_data);
	}while(0);
#endif
	return;
}
inline void mDisplayDataCC(hipfftComplex *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
	do
	{
		int maxlength = 25;
		cufftComplex *h_data;
		length = (length<=maxlength) ? length : maxlength;
		char MessageString[1000];
		h_data = (cufftComplex*)malloc(length * sizeof (cufftComplex));
		M_SAFE_CALL(cudaMemcpy(h_data, d_data, length*sizeof(cufftComplex), cudaMemcpyDeviceToHost));
		sprintf(MessageString,	"Line: %d\nData: ", line);
		for (int ii = 0;ii<length;++ii)
		{
			sprintf(MessageString,	"%s re: %f im: %f", MessageString, h_data[ii].x, h_data[ii].y);
		}
		//AfxMessageBox(MessageString, MB_ICONINFORMATION);
		free(h_data);
	}while(0);
#endif
	return;
}
inline void mDisplayDataUC(uint16_t *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
	do
	{
		int maxlength = 50;
		uint16_t *h_data;
		length = (length<=maxlength) ? length : maxlength;
		char MessageString[1000];
		h_data = (uint16_t*)malloc(length * sizeof (uint16_t));
		M_SAFE_CALL(cudaMemcpy(h_data, d_data, length*sizeof(uint16_t), cudaMemcpyDeviceToHost));
		sprintf(MessageString,	"Line: %d\nData: ", line);
		for (int ii = 0;ii<length;++ii)
		{
			sprintf(MessageString,	"%s %hhu", MessageString, h_data[ii]);
		}
		//AfxMessageBox(MessageString, MB_ICONINFORMATION);
		free(h_data);
	}while(0);
#endif
	return;
}
inline void mDisplayDataI(int *d_data, int length, int line)
{
#ifdef M_CUDA_DEBUG
	do
	{
		int maxlength = 50;
		int *h_data;
		length = (length<=maxlength) ? length : maxlength;
		char MessageString[1000];
		h_data = (int*)malloc(length * sizeof (int));
		M_SAFE_CALL(cudaMemcpy(h_data, d_data, length*sizeof(int), cudaMemcpyDeviceToHost));
		sprintf(MessageString,	"Line: %d\nData: ", line);
		for (int ii = 0;ii<length;++ii)
		{
			sprintf(MessageString,	"%s %d", MessageString, h_data[ii]);
		}
		//AfxMessageBox(MessageString, MB_ICONINFORMATION);
		free(h_data);
	}while(0);
#endif
	return;
}

//Calculate amplitudes in positions given by x, y, and z from a given hologram
extern "C" __declspec(dllexport)  int GetIandPhase(float *x_spots, float *y_spots, float *z_spots, float *h_pSLM_uc, int N_spots_all, int data_w, float *h_I_obt, float *h_Phase_obt)
{
	float *d_Iobtained_all;
	float *d_Pobtained_all;
	hipMalloc((void**)&d_Iobtained_all, N_spots_all*sizeof(float) );
	hipMalloc((void**)&d_Pobtained_all, N_spots_all*sizeof(float) );
	hipMemcpy(d_pSLM_uc, h_pSLM_uc, memsize_SLMuc, hipMemcpyHostToDevice);
	int offset = 0;
	int N_spots_rem = N_spots_all;
	int N_spots_this;
	while (N_spots_rem > 0)
	{
		N_spots_this = (N_spots_rem > MAX_SPOTS) ? MAX_SPOTS : N_spots_rem;
		hipMemcpyToSymbol(HIP_SYMBOL(c_x), x_spots+offset, N_spots_this*sizeof(float), 0, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(c_y), y_spots+offset, N_spots_this*sizeof(float), 0, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(c_z), z_spots+offset, N_spots_this*sizeof(float), 0, hipMemcpyHostToDevice);
		calculateIandPhase<<< N_spots_this, 512>>>(d_pSLM_uc, d_Iobtained_all+offset, d_Pobtained_all+offset);
		//calculateIobtained(uint16_t *g_pSLM_uc, float *g_Iobtained)
		hipDeviceSynchronize();
		N_spots_rem -= MAX_SPOTS;
		offset += MAX_SPOTS;
	}
	hipMemcpy(h_I_obt, d_Iobtained_all, N_spots_all*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_Phase_obt, d_Pobtained_all, N_spots_all*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_Iobtained_all);
	hipFree(d_Pobtained_all);
	status = hipGetLastError();
	return status;
}
